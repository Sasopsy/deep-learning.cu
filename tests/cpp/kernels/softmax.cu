#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <cassert>
#include <map>
#include <vector>
#include "csrc/utils/utils.hpp"
#include "csrc/kernels/softmax/launch.cuh"
#include "../test_utils.hpp"

namespace ck = cuda_kernel;

void checkResults(const float* cpu_result, const float* gpu_result, int N, int C) {
    bool passed = compareResults(cpu_result, gpu_result, N * C, 1e-5f, true);
    if (passed) {
        std::cout << "\033[32mResults match within tolerance!\033[0m" << std::endl;
    } else {
        std::cout << "\033[31mResults don't match within tolerance!\033[0m" << std::endl;
    }
}

void measureKernelTime(float* d_output, const float* d_input, int N, int C, int kernel_choice) {
    auto kernel = [&]() {
        ck::softmax::launch<float>(d_output, d_input, N, C, kernel_choice);
    };
    
    float ms = measureKernelPerformance(kernel);
    std::cout << "Kernel " << kernel_choice << " average execution time: " << ms << " ms" << std::endl;
}

void displayKernelChoices() {
    std::cout << "Kernel Choices:" << std::endl;
    std::cout << "0: Naive Kernel" << std::endl;
    std::cout << "1: Shared Memory Optimized Kernel" << std::endl;
    std::cout << "2: Intra Warp Optimized Kernel" << std::endl;
}

int getUserKernelChoice() {
    int choice;
    std::cout << "Enter the kernel choice: ";
    std::cin >> choice;
    return choice;
}

void printParams(int N, int C, int kernel_choice) {
    std::cout << "Parameters used:" << std::endl;
    std::cout << "N: " << N << std::endl;
    std::cout << "C: " << C << std::endl;
    std::cout << "Kernel Choice: " << kernel_choice << std::endl;
    std::cout << "Macros: " << std::endl;
    std::cout << "SOFTMAX_DEFAULT_SMEM_SIZE: " << SOFTMAX_DEFAULT_SMEM_SIZE << std::endl;
    std::cout << "SOFTMAX_DEFAULT_IWARP_SIZE: " << SOFTMAX_DEFAULT_IWARP_SIZE << std::endl;
}

void handleError(const std::string& error_message) {
    std::cerr << "Error: " << error_message << std::endl;
    exit(EXIT_FAILURE);
}

int getUserChoice() {
    int choice;
    std::cout << "Enter 0 to test all kernels, or 1 to choose a specific kernel: ";
    std::cin >> choice;
    return choice;
}

void testAllKernels(const std::vector<std::map<std::string, int>>& configurations) {
    for (const auto& config : configurations) {
        int N = config.at("N");
        int C = config.at("C");

        std::cout << "Testing configuration: N=" << N << ", C=" << C << std::endl;

        // Allocate memory and initialize data
        HostDevice<float> input(N * C);
        HostDevice<float> output(N * C);
        HostDevice<float> cpu_output(N * C);

        fillRandom(input.host, N * C);

        input.copyToDevice();

        for (int kernel_choice = 0; kernel_choice < 3; ++kernel_choice) {
            std::cout << "Testing kernel choice: " << kernel_choice << std::endl;

            // Launch the selected kernel
            ck::softmax::launch<float>(output.device, input.device, N, C, kernel_choice);

            // Copy results back to host
            output.copyToHost();

            // Compute reference result on CPU
            ck::softmax::cpu(cpu_output.host, input.host, N, C);

            // Check results
            checkResults(cpu_output.host, output.host, N, C);

            // Measure kernel performance
            measureKernelTime(output.device, input.device, N, C, kernel_choice);

            // Print parameters used
            printParams(N, C, kernel_choice);

            std::cout << " " << std::endl;
            
            // Clear device memory
            hipDeviceSynchronize();
            hipDeviceReset();
            
            // Reallocate and copy data back if we're not on the last iteration
            if (kernel_choice < 2) {
                input = HostDevice<float>(N * C);
                output = HostDevice<float>(N * C);
                fillRandom(input.host, N * C);
                input.copyToDevice();
            }
        }
    }
}

void testSpecificKernel(const std::vector<std::map<std::string, int>>& configurations, int kernel_choice) {
    for (const auto& config : configurations) {
        int N = config.at("N");
        int C = config.at("C");

        std::cout << "Testing configuration: N=" << N << ", C=" << C << std::endl;

        // Allocate memory and initialize data
        HostDevice<float> input(N * C);
        HostDevice<float> output(N * C);
        HostDevice<float> cpu_output(N * C);

        fillRandom(input.host, N * C);

        input.copyToDevice();

        std::cout << "Testing kernel choice: " << kernel_choice << std::endl;

        // Launch the selected kernel
        ck::softmax::launch<float>(output.device, input.device, N, C, kernel_choice);

        // Copy results back to host
        output.copyToHost();

        // Compute reference result on CPU
        ck::softmax::cpu(cpu_output.host, input.host, N, C);

        // Check results
        checkResults(cpu_output.host, output.host, N, C);

        // Measure kernel performance
        measureKernelTime(output.device, input.device, N, C, kernel_choice);

        // Print parameters used
        printParams(N, C, kernel_choice);

        std::cout << " " << std::endl;
        
        // Clear device memory after each configuration test
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

int main() {
    // List of dictionaries with different choices for N and C
    std::vector<std::map<std::string, int>> configurations = {
        {{"N", 8*12*128}, {"C", 64}}
    };

    // Get user choice
    int user_choice = getUserChoice();

    if (user_choice == 0) {
        // Test all kernels with different configurations
        testAllKernels(configurations);
    } else {
        // Display kernel choices and get user selection
        displayKernelChoices();
        int kernel_choice = getUserKernelChoice();
        testSpecificKernel(configurations, kernel_choice);
    }

    return 0;
}
